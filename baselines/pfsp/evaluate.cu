#include "hip/hip_runtime.h"
extern "C" {
#include "evaluate.h"
#include "stdlib.h" 
#include "lib/c_bound_simple_gpu_cuda.cu"
  //#include "lib/c_bound_johnson_gpu_cuda.cu"

  __device__ void swap_cuda(int* a, int* b)
  {
    int tmp = *b;
    *b = *a;
    *a = tmp;
  }

  void printDims(dim3 gridDim, dim3 blockDim) {
    printf("Grid Dimensions : [%d, %d, %d] blocks. \n",
    gridDim.x, gridDim.y, gridDim.z);

    printf("Block Dimensions : [%d, %d, %d] threads.\n",
    blockDim.x, blockDim.y, blockDim.z);
}

  // Evaluate a bulk of parent nodes on GPU using lb1
  __global__ void evaluate_gpu_lb1 (const int jobs, const int size, Node* parents_d, const lb1_bound_data* const lbound1_d, int* bounds)
  {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("thread Id: %d\n", threadId);
    if (threadId < size) {
      const int parentId = threadId / jobs; 
      const int k = threadId % jobs; 
      Node parent = parents_d[parentId]; 
      const uint8_t depth = parent.depth;
  
      // We evaluate all permutations by varying index k from limit1 forward
      if (k >= parent.limit1+1) {
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
	//bounds[threadId] =
	lb1_bound_gpu(lbound1_d, parent.prmu, parent.limit1+1,jobs,&bounds[threadId]);
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
      }
    }
  }

  //Still need to solve lb1_d index
  /*
    NOTE: This lower bound evaluates all the children of a given parent at the same time.
    Therefore, the GPU loop is on the parent nodes and not on the children ones, in contrast
    to the other lower bounds.
  */
  // Evaluate a bulk of parent nodes on GPU using lb1_d.
  __global__ void evaluate_gpu_lb1_d(const int jobs, const int size, const int* best, Node* parents_d, const lb1_bound_data* const lbound1_d, int* bounds)
  {
    // How does the NOTE translates into CUDA indices for searching only the parent nodes?
    int parentId = blockIdx.x * blockDim.x + threadIdx.x; // How to manage the proper indices?
    // I think that here maybe we do not to run through the threads ? 
    if(parentId < size/jobs){ 
      Node parent = parents_d[parentId];
      //const uint8_t depth = parent.depth; //not needed
      //const int* prmu = parent.prmu;

      // Vector of integers of size MAX_JOBS
      int lb_begin[MAX_JOBS];
    
      lb1_children_bounds_gpu(lbound1_d, parent.prmu, parent.limit1, jobs, lb_begin);

      // Going through the children for each parent node ?
      for(int k = 0; k < jobs; k++) {
	if (k >= parent.limit1+1) {
	  const int job = parent.prmu[k];
	  bounds[parentId*jobs+k] = lb_begin[job];
	}
      }
    }
  }

  // Evaluate a bulk of parent nodes on GPU using lb2.
  __global__ void evaluate_gpu_lb2(const int jobs, const int size, int* best, Node* parents_d, const lb1_bound_data* const lbound1_d, const lb2_bound_data* const lbound2_d, int* bounds)
  {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < size) {
      const int parentId = threadId / jobs; 
      const int k = threadId % jobs; 
      Node parent = parents_d[parentId];
      const uint8_t depth = parent.depth;
  
      // We evaluate all permutations by varying index k from limit1 forward
      if (k >= parent.limit1+1) {
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
	bounds[threadId] = lb2_bound_gpu(lbound1_d, lbound2_d, parent.prmu, parent.limit1+1, jobs, *best);
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
      }
    }
  }


  void evaluate_gpu(const int jobs, const int lb, const int size, const int nbBlocks, int* best, const lb1_bound_data* const lbound1, const lb2_bound_data* const lbound2, Node* parents, int* bounds)
  {
    // 1D grid of 1D blocks
    dim3 gridDim(nbBlocks);      // nbBlocks blocks in x direction, y, z default to 1
    dim3 blockDim(BLOCK_SIZE);     // BLOCK_SIZE threads per block in x direction
    // printDims(gridDim, blockDim);
    switch (lb) {
    case 0: // lb1_d
      evaluate_gpu_lb1_d<<<gridDim, blockDim>>>(jobs, size, best, parents, lbound1, bounds);
      return;
      break;

    case 1: // lb1
      evaluate_gpu_lb1<<<gridDim, blockDim>>>(jobs, size, parents, lbound1, bounds);
      return;
      break;

    case 2: // lb2
      evaluate_gpu_lb2<<<gridDim, blockDim>>>(jobs, size, best, parents, lbound1, lbound2, bounds);
      return;
      break;
    }
  }
}
