#include "hip/hip_runtime.h"
extern "C" {
#include "evaluate.h"
#include "stdlib.h" 
#include "lib/c_bound_simple_gpu_cuda.cu"
  //#include "lib/c_bound_johnson_gpu_cuda.cu"

  __device__ void swap_cuda(int* a, int* b)
  {
    int tmp = *b;
    *b = *a;
    *a = tmp;
  }

  // Evaluate a bulk of parent nodes on GPU using lb1
  __global__ void evaluate_gpu_lb1 (const int jobs, const int size, Node* parents_d, const lb1_bound_data* const lbound1_d, int* bounds)
  {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("thread Id: %d\n", threadId);

    if (threadId < size) {
      const int parentId = threadId / jobs; 
      const int k = threadId % jobs; 
      Node parent = parents_d[parentId]; 
      const uint8_t depth = parent.depth;
  
      // We evaluate all permutations by varying index k from limit1 forward
      if (k >= parent.limit1+1) {
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
	bounds[threadId] = lb1_bound_gpu(lbound1_d, parent.prmu, parent.limit1+1,jobs);
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
      }
    }
  }

  //Still need to solve lb1_d index
  /*
    NOTE: This lower bound evaluates all the children of a given parent at the same time.
    Therefore, the GPU loop is on the parent nodes and not on the children ones, in contrast
    to the other lower bounds.
  */
  // Evaluate a bulk of parent nodes on GPU using lb1_d.
  __global__ void evaluate_gpu_lb1_d(const int jobs, const int size, const int* best, Node* parents_d, const lb1_bound_data* const lbound1_d, int* bounds)
  {
    // How does the NOTE translates into CUDA indices for searching only the parent nodes?
    int parentId = blockIdx.x * blockDim.x + threadIdx.x; // How to manage the proper indices?
    // I think that here maybe we do not to run through the threads ? 
    if(parentId < size/jobs){ 
      Node parent = parents_d[parentId];
      //const uint8_t depth = parent.depth; //not needed
      //const int* prmu = parent.prmu;

      // Vector of integers of size MAX_JOBS
      int lb_begin[MAX_JOBS];
    
      lb1_children_bounds_gpu(lbound1_d, parent.prmu, parent.limit1, jobs, lb_begin);

      // Going through the children for each parent node ?
      for(int k = 0; k < jobs; k++) {
	if (k >= parent.limit1+1) {
	  const int job = parent.prmu[k];
	  bounds[parentId*jobs+k] = lb_begin[job];
	}
      }
    }
  }

  // Evaluate a bulk of parent nodes on GPU using lb2.
  __global__ void evaluate_gpu_lb2(const int jobs, const int size, int* best, Node* parents_d, const lb1_bound_data* const lbound1_d, const lb2_bound_data* const lbound2_d, int* bounds)
  {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < size) {
      const int parentId = threadId / jobs; 
      const int k = threadId % jobs; 
      Node parent = parents_d[parentId];
      const uint8_t depth = parent.depth;
  
      // We evaluate all permutations by varying index k from limit1 forward
      if (k >= parent.limit1+1) {
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
	//bounds[threadId] = lb2_bound_gpu(lbound1_d, lbound2_d, parent.prmu, parent.limit1+1, jobs, *best);
	swap_cuda(&parent.prmu[depth],&parent.prmu[k]);
      }
    }
  }


  void evaluate_gpu(const int jobs, const int lb, const int size, const int nbBlocks, const int numBounds, int* best,
		    const lb1_bound_data* const lbound1, const lb2_bound_data* const lbound2, Node* parent, int* bounds)
  {
    switch (lb) {
    case 0: // lb1_d
      evaluate_gpu_lb1_d<<<nbBlocks, BLOCK_SIZE>>>(jobs, size, best, parent, lbound1, bounds);
      return;
      break;

    case 1: // lb1
      evaluate_gpu_lb1<<<nbBlocks, BLOCK_SIZE>>>(jobs, size, parent, lbound1, bounds);
      return;
      break;

    case 2: // lb2
      evaluate_gpu_lb2<<<nbBlocks, BLOCK_SIZE>>>(jobs, size, best, parent, lbound1, lbound2, bounds);
      return;
      break;
    }
  }
}
